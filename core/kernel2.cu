#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include "Defines.h"

using namespace MonkeyGL;

typedef struct {
	float3 m[3];
} float3x3;

hipTextureObject_t volumeText;
hipArray* d_volumeArray = 0;

__constant__ hipTextureObject_t constTransferFuncTexts[MAXOBJECTCOUNT+1];
hipTextureObject_t transferFuncTexts[MAXOBJECTCOUNT+1];
hipArray *d_transferFuncArrays[MAXOBJECTCOUNT+1];

__constant__ float3 constAlphaAndWWWL[MAXOBJECTCOUNT+1];
float3 alphaAndWWWL[MAXOBJECTCOUNT+1];

hipTextureObject_t maskText;
hipArray* d_maskArray = 0;

float3 m_f3Nor, m_f3Spacing, m_f3maxper;
VOI m_voi;
hipExtent m_volumeSize;

__constant__ float3x3 constTransposeTransformMatrix;
__constant__ float3x3 constTransformMatrix;

unsigned char* d_pVR = 0;
int nWidth_VR = 0;
int nHeight_VR = 0;
short* d_pMPR = 0;
int nWidth_MPR = 0;
int nHeight_MPR = 0;

extern "C"
void cu_copyVolumeData( short* h_volumeData, hipExtent volumeSize)
{
	m_volumeSize = make_hipExtent(volumeSize.width, volumeSize.height, volumeSize.depth);

	if (d_volumeArray != 0)
	{
		checkCudaErrors(hipFreeArray(d_volumeArray));
		d_volumeArray = 0;
		volumeText = 0;
	}
	if (d_maskArray != 0)
	{
		checkCudaErrors(hipFreeArray(d_maskArray));
		d_maskArray = 0;
		maskText = 0;
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<short>();
	checkCudaErrors( hipMalloc3DArray(&d_volumeArray, &channelDesc, m_volumeSize) );

	hipMemcpy3DParms copyParams = {0};
	copyParams.dstArray = d_volumeArray;
	copyParams.extent   = m_volumeSize;
	copyParams.kind     = hipMemcpyHostToDevice;
	copyParams.srcPtr   = make_hipPitchedPtr(
		(void*)h_volumeData,
		m_volumeSize.width*sizeof(short),
		m_volumeSize.width,
		m_volumeSize.height
	);

	checkCudaErrors( hipMemcpy3D(&copyParams) );

	hipResourceDesc texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));

	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array = d_volumeArray;

	hipTextureDesc texDescr;
	memset(&texDescr, 0, sizeof(hipTextureDesc));

	texDescr.normalizedCoords = true;  // access with normalized texture coordinates
	texDescr.filterMode = hipFilterModeLinear;  // linear interpolation

	texDescr.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
	texDescr.addressMode[1] = hipAddressModeClamp;
	texDescr.addressMode[2] = hipAddressModeClamp;

	texDescr.readMode = hipReadModeNormalizedFloat;

	checkCudaErrors( hipCreateTextureObject(&volumeText, &texRes, &texDescr, NULL) );
}

extern "C"
void cu_InitCommon(float fxSpacing, float fySpacing, float fzSpacing)
{
	d_pVR = 0;
	nWidth_VR = 0;
	nHeight_VR = 0;

	m_f3Spacing.x = fxSpacing;
	m_f3Spacing.y = fySpacing;
	m_f3Spacing.z = fzSpacing;
	m_f3Nor.x = 1.0f / m_volumeSize.width;
	m_f3Nor.y = 1.0f / m_volumeSize.height;
	m_f3Nor.z = 1.0f / m_volumeSize.depth;
	float fMaxSpacing = max(fxSpacing, max(fySpacing, fzSpacing));

	float fMaxLen = max(m_volumeSize.width*fxSpacing, max(m_volumeSize.height*fySpacing, m_volumeSize.depth*fzSpacing));
	m_f3maxper.x = 1.0f*fMaxLen/(m_volumeSize.width*fxSpacing);
	m_f3maxper.y = 1.0f*fMaxLen/(m_volumeSize.height*fySpacing);
	m_f3maxper.z = 1.0f*fMaxLen/(m_volumeSize.depth*fzSpacing);

    for (int i=0; i<MAXOBJECTCOUNT; i++){
        d_transferFuncArrays[i] = 0;
    }
}

extern "C"
bool cu_setTransferFunc( float* pTransferFunc, int nLenTransferFunc, unsigned char nLabel)
{
	if (nLabel >= MAXOBJECTCOUNT){
		return false;
	}

	hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeArray;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));
    texDescr.normalizedCoords = true;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeElementType;

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();

    if (d_transferFuncArrays[nLabel] != 0)
	{
		checkCudaErrors(hipFreeArray(d_transferFuncArrays[nLabel]));
		d_transferFuncArrays[nLabel] = 0;
	}
    checkCudaErrors(hipMallocArray( &d_transferFuncArrays[nLabel], &channelDesc, nLenTransferFunc, 1));
    checkCudaErrors(
        hipMemcpy2DToArray(
            d_transferFuncArrays[nLabel],
            0,
            0,
            pTransferFunc,
            0,
            nLenTransferFunc*sizeof(float4),
            1,
            cudaMemcp1yHostToDevice
        )
    );

    texRes.res.array.array = d_transferFuncArrays[nLabel];

    hipTextureObject_t text = 0;
    checkCudaErrors(
        hipCreateTextureObject(&text, &texRes, &texDescr, NULL)
    );

    transferFuncTexts[nLabel] = text;
    hipMemcpyToSymbol(HIP_SYMBOL(constTransferFuncTexts), transferFuncTexts, sizeof(transferFuncTexts));

    return true;
}

extern "C"
void cu_copyOperatorMatrix( float *pTransformMatrix, float *pTransposeTransformMatrix)
{
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(constTransformMatrix), pTransformMatrix, sizeof(float3)*3) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(constTransposeTransformMatrix), pTransposeTransformMatrix, sizeof(float3)*3) );
}

extern "C"
void cu_copyAlphaAndWWWL(float *pAlphaAndWWWL)
{
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(constAlphaAndWWWL), pAlphaAndWWWL, sizeof(float3)*MAXOBJECTCOUNT+1) );
}

extern "C"
void cu_setVOI(VOI voi)
{
	m_voi.left = voi.left;
	m_voi.right = voi.right;
	m_voi.anterior = voi.anterior;
	m_voi.posterior = voi.posterior;
	m_voi.head = voi.head;
	m_voi.foot = voi.foot;
}

__device__ float3 mul(const float3x3 &M, const float3 &v)
{
	float3 r;
	r.x = dot(v, M.m[0]);
	r.y = dot(v, M.m[1]);
	r.z = dot(v, M.m[2]);
	return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
	rgba.x = __saturatef(rgba.x);
	rgba.y = __saturatef(rgba.y);
	rgba.z = __saturatef(rgba.z);
	rgba.w = 0.0f;
	return (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

__device__ float4 tracing(
	float4 sum,
	float alphaAccObject,
	hipTextureObject_t volumeText,
	float3 pos,
	float4 col,
	float3 dirLight,
	float3 f3Nor,
	bool invertZ
)
{
	float3 N;
	N.x = tex3D<float>(volumeText, pos.x+f3Nor.x, pos.y, pos.z) - tex3D<float>(volumeText, pos.x-f3Nor.x, pos.y, pos.z);
	N.y = tex3D<float>(volumeText, pos.x, pos.y+f3Nor.y, pos.z) - tex3D<float>(volumeText, pos.x, pos.y-f3Nor.y, pos.z);
	N.z = tex3D<float>(volumeText, pos.x, pos.y, pos.z+f3Nor.z) - tex3D<float>(volumeText, pos.x, pos.y, pos.z-f3Nor.z);
	if (invertZ){
		N.z = -N.z;
	}
	N = normalize(N);

	float diffuse = dot(N, dirLight);
	float4 clrLight = col * 0.35f;

	float4 f4Temp = make_float4(0.0f);
	if ( diffuse > 0.0f )
	{
		f4Temp = col * (diffuse*0.8f + 0.16f*(pow(diffuse, 8.0f)));
	}
	clrLight += f4Temp;

	diffuse = (1.0f - alphaAccObject) * col.w;
	return (sum + diffuse * clrLight);
}

__device__ bool getNextStep(
	float& fAlphaTemp,
	float& fStepTemp,
	float& accuLength,
	float fAlphaPre,
	float fStepL1,
	float fStepL4,
	float fStepL8
)
{
	if (fStepTemp == fStepL4)
		fAlphaTemp = 1 - pow(1-fAlphaTemp, 0.25f);
	else if(fStepTemp == fStepL8)
		fAlphaTemp = 1 - pow(1-fAlphaTemp, 0.125f);

	if (accuLength > 0.0f)
	{
		if (MAX(fAlphaTemp, fAlphaPre) > 0.001f)
		{
			if (fStepTemp == fStepL1)
			{
				accuLength -= (fStepL1 - fStepL4);
				fStepTemp = fStepL4;
				return false;
			}
			else if(fStepTemp == fStepL4)
			{
				accuLength -= (fStepL4 - fStepL8);
				fStepTemp = fStepL8;
				return false;
			}
		}
		else
		{
			if (fStepTemp == fStepL8)
				fStepTemp = fStepL4;
			else
				fStepTemp = fStepL1;
		}
	}
	return true;
}

/*
**   z
**   |__x
**  /-y
*/

__global__ void d_render(
	unsigned char* pPixelData,
	hipTextureObject_t volumeText,
	hipTextureObject_t maskText,
	int width,
	int height,
	float xTranslate,
	float yTranslate,
	float scale,
	float3 f3maxper,
	float3 f3Spacing,
	float3 f3Nor,
	VOI voi,
	hipExtent volumeSize,
	bool invertZ,
	float4 f4ColorBG
)
{
	const int x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	const int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

	if ((x < width) && (y < height) && (x >= 0) && (y >= 0))
	{
		uint nIdx = __umul24(y, width) + x;

		float u = 1.0f*(x-width/2.0f-xTranslate)/width;
		float v = 1.0f*(y-height/2.0f-yTranslate)/height;

		float4 sum = make_float4(0.0f);

		float3 dirLight = make_float3(0.0f, 1.0f, 0.0f);
		dirLight = normalize(mul(constTransformMatrix, dirLight));

		float fStepL1 = 1.0f/volumeSize.depth;
		float fStepL4 = fStepL1/4.0f;
		float fStepL8 = fStepL1/8.0f;
		float fStepTemp = fStepL1;

		float temp = 0.0f;
		float3 pos;

		float alphaAccObject[MAXOBJECTCOUNT+1];
		for (int i=0; i<MAXOBJECTCOUNT+1; i++){
			alphaAccObject[i] = 0.0f;
		}
		float alphaAcc = 0.0f;

		float accuLength = 0.0f;
		int nxIdx = 0;
		int nyIdx = 0;
		int nzIdx = 0;
		float fy = 0;

		float4 col;
		float fAlphaTemp = 0.0f;
		float fAlphaPre = 0.0f;

		unsigned char label = 0;
		float3 alphawwwl = make_float3(0.0f, 0.0f, 0.0f);

		while (accuLength < 1.732)
		{
			fy = (accuLength-0.866)*scale;

			pos = make_float3(u, fy, v);
			pos = mul(constTransformMatrix, pos);

			pos.x = pos.x * f3maxper.x + 0.5f;
			pos.y = pos.y * f3maxper.y + 0.5f;
			pos.z = pos.z * f3maxper.z + 0.5f;
			if (invertZ)
				pos.z = 1.0f - pos.z;

			nxIdx = pos.x * volumeSize.width;
			nyIdx = pos.y * volumeSize.height;
			nzIdx = pos.z * volumeSize.depth;
			if (nxIdx<voi.left || nxIdx>voi.right || nyIdx<voi.posterior || nyIdx>voi.anterior || nzIdx<voi.head || nzIdx>voi.foot)
			{
				accuLength += fStepTemp;
				continue;
			}
			if(maskText == 0){
				label = 0;
			}
			else {
				label = tex3D<unsigned char>(maskText, nxIdx, nyIdx, nzIdx);
			}
			alphawwwl = constAlphaAndWWWL[label];

			temp = 32768*tex3D<float>(volumeText, pos.x, pos.y, pos.z);
			temp = (temp - alphawwwl.z)/alphawwwl.y + 0.5;
			if (temp>1)
				temp = 1;

			col = tex1D<float4>(constTransferFuncTexts[label], temp);

			fAlphaTemp = col.w;

			if (!getNextStep(fAlphaTemp, fStepTemp, accuLength, fAlphaPre, fStepL1, fStepL4, fStepL8)){
				continue;
			}

			fAlphaPre = fAlphaTemp;
			accuLength += fStepTemp;

			col.w = fAlphaTemp;

			if (col.w > 0.0005f && alphaAccObject[label] < alphawwwl.x){
				sum = tracing(sum, alphaAcc, volumeText, pos, col, dirLight, f3Nor, invertZ);
				alphaAccObject[label] += (1.0f - alphaAcc) * col.w;
				alphaAcc += (1.0f - alphaAcc) * col.w;
			}

			if (alphaAcc > 0.995f){
				break;
			}

		}

		if (sum.x==0.0f && sum.y==0.0f && sum.z==0.0f && sum.w==0.0f){
			sum = f4ColorBG;
		}

		unsigned int result = rgbaFloatToInt(sum);

		pPixelData[nIdx*3]	 = result & 0xFF; //R
		pPixelData[nIdx*3+1] = (result>>8) & 0xFF; //G
		pPixelData[nIdx*3+2] = (result>>16) & 0xFF; //B
	}
}

extern "C"
void cu_render(unsigned char* pVR, int width, int height, float xTranslate, float yTranslate, float scale, bool invertZ, RGBA colorBG)
{
	if (width>nWidth_VR || height>nHeight_VR)
	{
		if (d_pVR != 0)
			checkCudaErrors(hipFree(d_pVR));
		nWidth_VR = width;
		nHeight_VR = height;
		checkCudaErrors(hipMalloc( (void**)&d_pVR, nWidth_VR*nHeight_VR*3*sizeof(unsigned char) ));
	}

	dim3 blockSize(32, 32);
	dim3 gridSize( (width-1)/blockSize.x+1, (height-1)/blockSize.y+1 );

	float4 clrBG = make_float4(colorBG.red, colorBG.green, colorBG.blue, colorBG.alpha);

	d_render<<<gridSize, blockSize>>>(
		d_pVR,
		volumeText,
		maskText,
		width,
		height,
		xTranslate,
		yTranslate,
		scale,
		m_f3maxper,
		m_f3Spacing,
		m_f3Nor,
		m_voi,
		m_volumeSize,
		invertZ,
		clrBG
	);
	hipError_t t = hipMemcpy( pVR, d_pVR, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost );
}
